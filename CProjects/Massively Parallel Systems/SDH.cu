/* ==================================================================
	Programmer: Yicheng Tu (ytu@cse.usf.edu)
  Modified By: Garrett Moncrief
	The basic SDH algorithm implementation for 3D data
	To compile: nvcc SDH.c -o SDH in the c4cuda machines
   ==================================================================
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>


#define BOX_SIZE	23000 /* size of the data box on one dimension            */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	double x_pos;
	double y_pos;
	double z_pos;
} atom;

typedef struct hist_entry{
	//float min;
	//float max;
	long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;


bucket * histogram, * histogram2;		/* list of all buckets in the histogram   */
long long	PDH_acnt;	/* total number of data points            */
int num_buckets;		/* total number of buckets in the histogram */
double   PDH_res;		/* value of w                             */
atom * atom_list;		/* list of all data points                */

/* These are for an old way of tracking time */
struct timezone Idunno;	
struct timeval startTime, endTime;


/* 
	distance of two points in the atom_list for the CPU side
*/
double p2p_distance(int ind1, int ind2) {
	
	double x1 = atom_list[ind1].x_pos;
	double x2 = atom_list[ind2].x_pos;
	double y1 = atom_list[ind1].y_pos;
	double y2 = atom_list[ind2].y_pos;
	double z1 = atom_list[ind1].z_pos;
	double z2 = atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


/* 
	distance of two points in the atom_list for the GPU side 
*/
__device__
double p2p_distance_GPU(atom* d_atom_list, int ind1, int ind2) {
	
	double x1 = d_atom_list[ind1].x_pos;
	double x2 = d_atom_list[ind2].x_pos;
	double y1 = d_atom_list[ind1].y_pos;
	double y2 = d_atom_list[ind2].y_pos;
	double z1 = d_atom_list[ind1].z_pos;
	double z2 = d_atom_list[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1-x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


/* 
	brute-force SDH solution in a single CPU thread 
*/
int PDH_baseline() {
	int i, j, h_pos;
	double dist;
	
	for(i = 0; i < PDH_acnt; i++) {
		for(j = i+1; j < PDH_acnt; j++) {
			dist = p2p_distance(i,j);
			h_pos = (int) (dist / PDH_res);
			histogram[h_pos].d_cnt++;
		} 
	}
	return 0;
}

/* Implementation for GPU SDH function */
__global__
void GPU_baseline(bucket* d_histogram, atom* d_atom_list, double PDH_res, int PDH_acnt)
{
	//printf("\n###Hello from GPUn");
	int i= blockDim.x*blockIdx.x + threadIdx.x;
	int h_pos;
	//calculates unique global index
	//printf("i = %d \n", i);
	double dist;
	
	for(int j = i+1; j < PDH_acnt; j++) {
		dist = p2p_distance_GPU(d_atom_list, i,j);
		h_pos = (int) (dist / PDH_res);
		d_histogram[h_pos].d_cnt++;
	} 
}

/* 
	set a checkpoint and show the (natural) running time in seconds 
*/
double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("%ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}


/* 
	print the counts in all buckets of the histogram 
*/
void output_histogram(){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram[i].d_cnt);
		total_cnt += histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}

/* 
	print the counts in all buckets of the histogram 
*/
void output_histogram2(){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram2[i].d_cnt);
		total_cnt += histogram2[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}

void output_histogram_diff(bucket *histogram, bucket *histogram2){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram2[i].d_cnt - histogram[i].d_cnt);
		total_cnt += (histogram2[i].d_cnt - histogram[i].d_cnt);
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}



int main(int argc, char **argv)
{
	int i;

	PDH_acnt = atoi(argv[1]);
	PDH_res	 = atof(argv[2]);
//printf("args are %d and %f\n", PDH_acnt, PDH_res);

	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
	histogram = (bucket *)malloc(sizeof(bucket)*num_buckets);
	histogram2 = (bucket *)malloc(sizeof(bucket)*num_buckets);

	atom_list = (atom *)malloc(sizeof(atom)*PDH_acnt);

	
	srand(1);
	/* generate data following a uniform distribution */
	for(i = 0;  i < PDH_acnt; i++) {
		atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
	}
	/* start counting time */
	//gettimeofday(&startTime, &Idunno);
	
	/* call CPU single thread version to compute the histogram */
	PDH_baseline();
	
	/* print out the histogram */
	printf("\nCPU brute-force histogram\n");
	output_histogram();
	printf("\nGPU histogram\n");
	
	/* 
		### run time check commented out per instruction to only include histogram
		and differences between histograms ###
	*/  
	//printf("\nRunning time for CPU version: ");
	//report_running_time();
	
	/* ### BEGIN GPU VERSION ### */
	
	/* start counting time */
	//gettimeofday(&startTime, &Idunno);
	
	/* Allocate memory as shown in chapter 2.4 */
	bucket* d_histogram;
	atom* d_atom_list;
	
	hipMalloc((void **) &d_histogram, sizeof(bucket)*num_buckets);
	hipMemcpy(d_histogram, histogram, sizeof(bucket)*num_buckets, hipMemcpyHostToDevice);
	
	hipMalloc((void **) &d_atom_list, sizeof(atom)*PDH_acnt);
	hipMemcpy(d_atom_list, atom_list, sizeof(atom)*PDH_acnt, hipMemcpyHostToDevice);
	
	/* 
		call GPU version to complete histogram using execution config params per 2.5
		where <<<number of thread blocks, number of threads in each block>>>
	*/
	
	GPU_baseline <<<ceil(PDH_acnt/32), 32>>> (d_histogram, d_atom_list, PDH_res, PDH_acnt);
	
	/* transfer memory back to host per page 31 */
	hipMemcpy(histogram2, d_histogram, sizeof(bucket)*num_buckets, hipMemcpyHostToHost);
	
	/* Free memory as per chapter 2.4*/
	hipFree(d_histogram);
	hipFree(d_atom_list);
	
	/* 
		### run time check commented out per instruction to only include histogram
		and differences between histograms ###
	*/ 
	//printf("\nRunning time for GPU version: ");
	//report_running_time();
	
	/* print out the histogram */
	output_histogram2();
	
	printf("\nDifference between GPU histogram - CPU histogram\n");
	output_histogram_diff(histogram, histogram2);
	
	return 0;
}


